#include "hip/hip_runtime.h"
#include <GL/glew.h>
#include <GL/freeglut.h>
#include <gl/GLU.h>
#include <GL/gl.h>

#include "hip/hip_runtime.h"
#include ""
#include <cuda_gl_interop.h>

#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <helper_cuda_gl.h>

#include <hip/hip_vector_types.h>

#include <stdio.h>
#include <math.h>

#include <Windows.h>

#define WINDOWS_MEAN_AND_LEAN
#define NOMINAX
#define EXPORT_API __declspec(dllexport)

static float g_Time;
static float g_Freq = 4.0f;

static float mesh_width;
static float mesh_height;

static GLuint vbo;
struct hipGraphicsResource *cuda_vbo_resource;
void *d_vbo_buffer = NULL;

extern "C" void EXPORT_API SetTimeFromUnity(float t) { g_Time = t; }

extern "C"  float EXPORT_API ComputeSineWave(const float u, const float v)
{
	return sin(u * g_Freq + g_Time) * cos(v * g_Freq + g_Time) * 0.5f;
}

__global__ void simple_vbo_kernel(float3 *pos, unsigned int width, unsigned int height, float time)
{
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	// calculate uv coordinates
	float u = x / (float)width;
	float v = y / (float)height;
	u = u*2.0f - 1.0f;
	v = v*2.0f - 1.0f;

	// calculate simple sine wave pattern
	float freq = 4.0f;
	float w = sinf(u*freq + time) * cosf(v*freq + time) * 0.5f;

	// write output vertex
	pos[y*width + x] = make_float3(u, w, v);
}

extern "C" float3* EXPORT_API ParallelComputeSineWave(float3* verts, float width, float height, float time)
{
	dim3 block(8, 8, 1);
	dim3 grid(width / block.x, height / block.y, 1);
	simple_vbo_kernel << < grid, block >> >(verts, width, height, time);

	return verts;
}


void launch_kernel(float3 *pos, unsigned int mesh_width, unsigned int mesh_height, float time)
{
	// execute the kernel
	dim3 block(8, 8, 1);
	dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
	simple_vbo_kernel << < grid, block >> >(pos, mesh_width, mesh_height, time);
}


void runCuda(struct hipGraphicsResource **vbo_resource)
{
	// map OpenGL buffer object for writing from CUDA
	float3 *dptr;
	checkCudaErrors(hipGraphicsMapResources(1, vbo_resource, 0));
	size_t num_bytes;
	checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&dptr, &num_bytes,
		*vbo_resource));
	//printf("CUDA mapped VBO: May access %ld bytes\n", num_bytes);

	// execute the kernel
	//    dim3 block(8, 8, 1);
	//    dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
	//    kernel<<< grid, block>>>(dptr, mesh_width, mesh_height, g_fAnim);

	launch_kernel(dptr, mesh_width, mesh_height, g_Time);

	// unmap buffer object
	checkCudaErrors(hipGraphicsUnmapResources(1, vbo_resource, 0));
}

void createVBO(GLuint *vbo, struct hipGraphicsResource **vbo_res, unsigned int vbo_res_flags)
{
	assert(vbo);

	// TODO this GL calls need to be moved to the C# script

	// create buffer object
	// glGenBuffers(1, vbo);
	// glBindBuffer(GL_ARRAY_BUFFER, *vbo);

	// initialize buffer object
	unsigned int size = mesh_width * mesh_height * 3 * sizeof(float);
	// glBufferData(GL_ARRAY_BUFFER, size, 0, GL_DYNAMIC_DRAW);

	// glBindBuffer(GL_ARRAY_BUFFER, 0);

	// register this buffer object with CUDA
	checkCudaErrors(hipGraphicsGLRegisterBuffer(vbo_res, *vbo, vbo_res_flags));
}

void deleteVBO(GLuint *vbo, struct hipGraphicsResource *vbo_res)
{

	// unregister this buffer object with CUDA
	checkCudaErrors(hipGraphicsUnregisterResource(vbo_res));

	// glBindBuffer(1, *vbo);
	// glDeleteBuffers(1, vbo);

	*vbo = 0;
}

void cleanup()
{
	if (vbo)
	{
		deleteVBO(&vbo, cuda_vbo_resource);
	}

	// hipDeviceReset causes the driver to clean up all state. While
	// not mandatory in normal operation, it is good practice.  It is also
	// needed to ensure correct operation when the application is being
	// profiled. Calling hipDeviceReset causes all profile data to be
	// flushed before the application exits
	hipDeviceReset();
}