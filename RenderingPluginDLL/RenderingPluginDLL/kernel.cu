#include "hip/hip_runtime.h"
#include <GL/glew.h>
#include <GL/freeglut.h>
#include <gl/GLU.h>
#include <GL/gl.h>

#include "hip/hip_runtime.h"
#include ""
#include <cuda_gl_interop.h>

#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <helper_cuda_gl.h>

#include <hip/hip_vector_types.h>

#include <stdio.h>
#include <math.h>

#include <Windows.h>

#define WINDOWS_MEAN_AND_LEAN
#define NOMINAX
#define EXPORT_API __declspec(dllexport)

typedef void(*FuncPtr)(const char *);
FuncPtr Debug;

static int texSize;

static int* triangles;
static int triangleCount;
static int* devTArray;

static struct hipGraphicsResource* cgr;
static void* texPtr;
static GLuint texID;

static struct hipGraphicsResource* nCgr;
static void* nTexPtr;
static GLuint nTexID;

static float unityTime;

extern "C" void EXPORT_API SetTimeFromUnity(float t) { unityTime = t; }


__global__ void plugin_kernel(hipSurfaceObject_t cso, hipSurfaceObject_t nCso, int meshSize, int* triangles, int trCount, float time)
{
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	float freq = 4.0f;
	float attenuation = 0.5f;

	float4 vert = surf2Dread<float4>(cso, (int)sizeof(float4)*x, y, hipBoundaryModeZero);
	vert.y = sinf(vert.x * freq + time) * cosf(vert.z * freq + time) * attenuation;

	surf2Dwrite(vert, cso, (int)sizeof(float4) * x, y, hipBoundaryModeZero);

	__syncthreads();

	int vertID = x * meshSize + y;

	float4 normal = { 0, 0, 0, 0 };

	for (int i = 0; i < trCount; i += 3)
	{
		if (triangles[i] == vertID || triangles[i + 1] == vertID || triangles[i + 2] == vertID)
		{
			int xx = triangles[i] / meshSize;
			int yy = triangles[i] % meshSize;

			float4 vertA = surf2Dread<float4>(cso, (int)sizeof(float4) *  xx, yy, hipBoundaryModeZero);

			xx = triangles[i + 1] / meshSize;
			yy = triangles[i + 1] % meshSize;

			float4 vertB = surf2Dread<float4>(cso, (int)sizeof(float4) *  xx, yy, hipBoundaryModeZero);

			xx = triangles[i + 2] / meshSize;
			yy = triangles[i + 2] % meshSize;

			float4 vertC = surf2Dread<float4>(cso, (int)sizeof(float4) *  xx, yy, hipBoundaryModeZero);

			float3 vecA = { vertB.x - vertA.x, vertB.y - vertA.y, vertB.z - vertA.z };
			float3 vecB = { vertC.x - vertA.x, vertC.y - vertA.y, vertC.z - vertA.z };

			float3 cross = { vecA.y * vecB.z - vecA.z * vecB.y, vecA.z * vecB.x - vecA.x * vecB.z, vecA.x * vecB.y - vecA.y * vecB.x };
			float lenCross = sqrt(cross.x * cross.x + cross.y * cross.y + cross.z * cross.z);

			if (lenCross > 0.0f)
			{
				cross.x /= lenCross;
				cross.y /= lenCross;
				cross.z /= lenCross;
			}

			normal.x += cross.x;
			normal.y += cross.y;
			normal.z += cross.z;
		}

		float lenNormal = sqrt(normal.x * normal.x + normal.y * normal.y + normal.z * normal.z);

		if (lenNormal > 0.0f)
		{
			normal.x /= lenNormal;
			normal.y /= lenNormal;
			normal.z /= lenNormal;
		}

		surf2Dwrite(normal, nCso, (int)sizeof(float4) * x, y, hipBoundaryModeZero);
	}
}

void CheckPluginErrors(hipError_t err, const char* context)
{
	if (err != hipSuccess)
	{
		const char* errName = hipGetErrorName(err);
		const char* errString = hipGetErrorString(err);

		char* errMessage = (char*)calloc(strlen(errName) + strlen(errString) + 8, sizeof(char));
		strcpy(errMessage, context);
		strcat(errMessage, " --> ");
		strcat(errMessage, errName);
		strcat(errMessage, ": ");
		strcat(errMessage, errString);

		Debug(errMessage);
	}
}

void UpdateVertsInTex()
{
	// Vertex data
	CheckPluginErrors(hipGraphicsMapResources(1, &cgr), "Error encountered while mapping resource");

	hipArray_t hipArray;
	CheckPluginErrors(hipGraphicsSubResourceGetMappedArray(&hipArray, cgr, 0, 0), "Error encountered while mapping graphics resource to CUDA array.");

	hipResourceDesc desc;
	desc.resType = hipResourceTypeArray;
	desc.res.array.array = hipArray;

	hipSurfaceObject_t cso;
	CheckPluginErrors(hipCreateSurfaceObject(&cso, &desc), "Error encountered while creating Surface Object.");

	// Normal data
	CheckPluginErrors(hipGraphicsMapResources(1, &nCgr), "Error encountered while mapping resource");

	hipArray_t nCudaArray;
	CheckPluginErrors(hipGraphicsSubResourceGetMappedArray(&nCudaArray, nCgr, 0, 0), "Error encountered while mapping graphics resource to CUDA array.");

	hipResourceDesc nDesc;
	nDesc.resType = hipResourceTypeArray;
	nDesc.res.array.array = nCudaArray;

	hipSurfaceObject_t nCso;
	CheckPluginErrors(hipCreateSurfaceObject(&nCso, &nDesc), "Error encountered while creating Surface Object.");

	dim3 block(texSize, texSize, 1);
	dim3 grid(texSize / block.x, texSize / block.y, 1);
	plugin_kernel << < grid, block >> >(cso, nCso, texSize, devTArray, triangleCount, unityTime);

	CheckPluginErrors(hipGetLastError(), "Error in kernel execution.");

	CheckPluginErrors(hipDestroySurfaceObject(cso), "Error encountered while destroying Surface Object.");
	CheckPluginErrors(hipGraphicsUnmapResources(1, &cgr), "Error encountered while unmapping resource.");

	CheckPluginErrors(hipDestroySurfaceObject(nCso), "Error encountered while destroying Surface Object.");
	CheckPluginErrors(hipGraphicsUnmapResources(1, &nCgr), "Error encountered while unmapping resource.");

	CheckPluginErrors(hipStreamSynchronize(0), "Error in stream synchronization.");
}

extern "C" void EXPORT_API UnityRenderEvent(int eventID)
{
	UpdateVertsInTex();
}

extern "C" EXPORT_API void SetDebugFunction(FuncPtr fp)
{
	Debug = fp;
}

extern "C" EXPORT_API void Init(int size, void* tPtr, void* nTPtr, int* tr, int trCount)
{
	texSize = size;

	triangles = tr;
	triangleCount = trCount;

	texPtr = tPtr;
	texID = (GLuint)(size_t)(texPtr);

	CheckPluginErrors(hipGraphicsGLRegisterImage(&cgr, texID, GL_TEXTURE_2D, cudaGraphicsMapFlagsNone), "Error encountered while registering resource.");

	nTexPtr = nTPtr;
	nTexID = (GLuint)(size_t)(nTexPtr);

	CheckPluginErrors(hipGraphicsGLRegisterImage(&nCgr, nTexID, GL_TEXTURE_2D, cudaGraphicsMapFlagsNone), "Error encountered while registering resource.");

	CheckPluginErrors(hipMalloc((void**)&devTArray, trCount * sizeof(int)), "Error encountered while allocating memory on device");
	CheckPluginErrors(hipMemcpy(devTArray, triangles, trCount * sizeof(int), hipMemcpyHostToDevice), "Error encountered while copying array to device");
}

extern "C" EXPORT_API void Cleanup()
{
	CheckPluginErrors(hipGraphicsUnregisterResource(cgr), "Error encountered while unregistering resource.");
	CheckPluginErrors(hipGraphicsUnregisterResource(nCgr), "Error encountered while unregistering resource.");

	CheckPluginErrors(hipFree(devTArray), "Error encountered while freeing memory on device");
}